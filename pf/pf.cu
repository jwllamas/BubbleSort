#include "hip/hip_runtime.h"
/*
 * CS 470 Research Project
 * Team BubbleSort
 */

#include <stdio.h>
#include <stdlib.h>

// Custom timing macros
#include "timer.h"

// Controls debug output
//#define DEBUG

// Greyscale maximum (Set to 3 rather than 255 so there is a 33% chance it is 0)
#define GMAX 3

// Size of image in pixels
#define SIZE 512

// Greyscale vals for each pixel
int *gscales;

// Distance vals for each pixel
int *dist;

int total_pixels;

// Create random gscale vals for each pixel
void rand_gscales()
{
    for (int x = 0; x < SIZE; x++)
    {
        for (int y = 0; y < SIZE; y++)
        {
            for (int z = 0; z < SIZE; z++)
            {
                int r = rand() % GMAX; // Random int 0-2 (inclusive)
		int index = (SIZE * SIZE) * x + SIZE * y + z;
                gscales[index] = r;  // Assign random int to gscale index
            }
        }
    }
}

// Debug output random gscale values
void debug_rand()
{
    for (int i  = 0; i < SIZE * 2; i++)
    {
        printf("-");
    }
   
    printf("\nRandom greyscale values:");
    for (int x = 0; x < SIZE; x++)
    {
        printf("\n");
        for (int y = 0; y < SIZE; y++)
        {
            printf("\n");
            for (int z = 0; z < SIZE; z++)
            {
		int index = (SIZE * SIZE) * x + SIZE * y + z;
                printf("%d ", gscales[index]);
            }
        }
    }
    printf("\n\nX val is each block");
    printf("\nY val is each row");
    printf("\nZ val is each column\n");
}


// Search for the nearest white pixel
__global__ void find_dist(int n, int *g, int *d)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        int x1 = i/(SIZE*SIZE);
        int y1 = (i%(SIZE*SIZE))/SIZE;
	int z1 = (i%(SIZE*SIZE))%SIZE;

        int index1 = (SIZE * SIZE) * x1 + SIZE * y1 + z1;
                
        if (g[index1] == 0)
        {
            d[index1] = 0;
        }
        else
        {
            int lowestDist = SIZE;
            for (int j = 0; j < n; j++) 
            {
                int x2 = j/(SIZE*SIZE);
                int y2 = (j%(SIZE*SIZE))/SIZE;
                int z2 = (j%(SIZE*SIZE))%SIZE;

	        int index2 = (SIZE * SIZE) * x2 + SIZE * y2 + z2;
                                
	        if (g[index2] == 0)
                {
                    int tmpDist = ((x2-x1)*(x2-x1)) + ((y2-y1)*(y2-y1)) + ((z2 - z1)*(z2-z1));
                    if (tmpDist < lowestDist)
                    {
                        lowestDist = tmpDist;
                    } 
                }
	    }

            d[index1] = lowestDist;
	}
    }
}

// Debug output random gscale values
void debug_dist()
{
    for (int i  = 0; i < SIZE * 2; i++)
    {
        printf("-");
    }

    printf("\nDistance values:");
    for (int x = 0; x < SIZE; x++)
    {
        printf("\n");
        for (int y = 0; y < SIZE; y++)
        {
            printf("\n");
            for (int z = 0; z < SIZE; z++)
            {
		int index = (SIZE * SIZE) * x + SIZE * y + z;
                printf("%d ", dist[index]);
            }
        }
    }
    printf("\n\nX val is each block");
    printf("\nY val is each row");
    printf("\nZ val is each column\n");
}

int main(int argc, char *argv[])
{
    FILE *file;
    file = fopen("output.txt", "w+");

    total_pixels = SIZE*SIZE*SIZE;
    hipMallocManaged(&gscales, (total_pixels * sizeof(int)));
    hipMallocManaged(&dist, (total_pixels * sizeof(int))); 
    
    rand_gscales(); // Allocate random gscale values to array

    #ifdef DEBUG
    printf("\nDebug Output\n");
    debug_rand();
    #endif 

    int blockSize = 1024;
    int numBlocks = total_pixels / blockSize;

    START_TIMER(findDist);

    find_dist<<< numBlocks, blockSize >>>(total_pixels, gscales, dist);
    hipDeviceSynchronize();

    STOP_TIMER(findDist);

    #ifdef DEBUG
    debug_dist();
    #endif

    fprintf(file, "DIST: %8.4fs\n", GET_TIMER(findDist));

    fclose(file);
    hipFree(gscales);
    hipFree(dist);
    return EXIT_SUCCESS;
}

